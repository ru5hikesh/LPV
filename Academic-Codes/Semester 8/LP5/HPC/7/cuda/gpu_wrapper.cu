/*
 * GPU wrapper implementations
 * GPU launch function implementation
 * loads the data on the device
 */

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "../include/color.h"
#include "../include/parallel.h"

#define BLOCK_SIZE 1024
#define DEBUG 1
#define NUM_BYTES 10240

__constant__ unsigned char device_bitSequenceConstMemory[256][255];

/*
 * Main launching function to load the data on the device
 */
void launchCudaHuffmanCompress(unsigned char *inputFileData, unsigned int *compressedDataOffset,
                               unsigned int inputFileLength, int numberOfKernels,
                               unsigned int integerOverflowFlag, long unsigned int memoryRequired) {
    struct huffmanDictionary *device_huffmanDictionary;
    unsigned char *device_inputFileData, *device_byteCompressedData;
    unsigned int *device_compressedDataOffset;
    unsigned int *gpuBitPaddingFlag, *bitPaddingFlag;
    unsigned int *gpuMemoryOverflowIndex, *integerOverflowIndex;
    long unsigned int memoryFree, memoryTotal;
    hipError_t error;

    // generating the offset
    // no integer overflow
    if (integerOverflowFlag == 0) {
        // single run no overflow
        if (numberOfKernels == 1) {
            createDataOffsetArray(compressedDataOffset, inputFileData, inputFileLength);
        }

        // multiple run with no overflow [big files]
        else {
            gpuBitPaddingFlag = (unsigned int *)calloc(numberOfKernels, sizeof(unsigned int));
            gpuMemoryOverflowIndex =
                (unsigned int *)calloc(numberOfKernels * 2, sizeof(unsigned int));
            createDataOffsetArray(compressedDataOffset, inputFileData, inputFileLength,
                                  gpuMemoryOverflowIndex, gpuBitPaddingFlag, memoryRequired);
        }
    }

    // integer overflow
    else {
        // single run overflow
        if (numberOfKernels == 1) {
            bitPaddingFlag = (unsigned int *)calloc(numberOfKernels, sizeof(unsigned int));
            integerOverflowIndex =
                (unsigned int *)calloc(numberOfKernels * 2, sizeof(unsigned int));
            createDataOffsetArray(compressedDataOffset, inputFileData, inputFileLength);
        }

        // multiple run overflow
        else {
            gpuBitPaddingFlag = (unsigned int *)calloc(numberOfKernels, sizeof(unsigned int));
            bitPaddingFlag = (unsigned int *)calloc(numberOfKernels, sizeof(unsigned int));
            integerOverflowIndex =
                (unsigned int *)calloc(numberOfKernels * 2, sizeof(unsigned int));
            gpuMemoryOverflowIndex =
                (unsigned int *)calloc(numberOfKernels * 2, sizeof(unsigned int));
            createDataOffsetArray(compressedDataOffset, inputFileData, inputFileLength,
                                  integerOverflowIndex, bitPaddingFlag, gpuMemoryOverflowIndex,
                                  gpuBitPaddingFlag, NUM_BYTES, memoryRequired);
        }
    }

    // gpu initiation
    {
        // memory allocation
        error = hipMalloc((void **)&device_inputFileData, inputFileLength * sizeof(unsigned char));
        if (error != hipSuccess)
            printf("\n%sError 1 :: %s", COLOR_ERROR, hipGetErrorString(error));

        error = hipMalloc((void **)&device_compressedDataOffset,
                           (inputFileLength + 1) * sizeof(unsigned int));
        if (error != hipSuccess) printf("\nError 2 :: %s", hipGetErrorString(error));

        error = hipMalloc((void **)&device_huffmanDictionary, sizeof(huffmanDictionary));
        if (error != hipSuccess) printf("\nError 3 :: %s", hipGetErrorString(error));

        // memory copy to device
        error = hipMemcpy(device_inputFileData, inputFileData,
                           inputFileLength * sizeof(unsigned char), hipMemcpyHostToDevice);
        if (error != hipSuccess) printf("\nError 4 :: %s", hipGetErrorString(error));

        error = hipMemcpy(device_compressedDataOffset, compressedDataOffset,
                           (inputFileLength + 1) * sizeof(unsigned int), hipMemcpyHostToDevice);
        if (error != hipSuccess) printf("\nError 5 :: %s", hipGetErrorString(error));

        error = hipMemcpy(device_huffmanDictionary, &huffmanDictionary, sizeof(huffmanDictionary),
                           hipMemcpyHostToDevice);
        if (error != hipSuccess) printf("\nError 6 :: %s", hipGetErrorString(error));

        // constant memory if required
        if (constMemoryFlag == 1) {
            error = hipMemcpyToSymbol(HIP_SYMBOL(device_bitSequenceConstMemory), bitSequenceConstMemory,
                                       265 * 255 * sizeof(unsigned char));
            if (error != hipSuccess) printf("\nError Constant :: %s", hipGetErrorString(error));
        }
    }

    // Single run
    if (numberOfKernels == 1) {
        // no overflow
        if (integerOverflowFlag == 0) {
            error = hipMalloc((void **)&device_byteCompressedData,
                               (compressedDataOffset[inputFileLength]) * sizeof(unsigned char));
            if (error != hipSuccess) printf("\nError 7 :: %s", hipGetErrorString(error));

            // initialize device_byteCompressedData
            error = hipMemset(device_byteCompressedData, 0,
                               compressedDataOffset[inputFileLength] * sizeof(unsigned char));
            if (error != hipSuccess) printf("\nError 8 :: %s", hipGetErrorString(error));

            // debug
            if (1) {
                hipMemGetInfo(&memoryFree, &memoryTotal);
                printf("\nFree Mem: %lu", memoryFree);
            }

            // run kernel
            compress<<<1, BLOCK_SIZE>>>(device_inputFileData, device_compressedDataOffset,
                                        device_huffmanDictionary, device_byteCompressedData,
                                        inputFileLength, constMemoryFlag);
            hipError_t error_kernel = hipGetLastError();
            if (error_kernel != hipSuccess)
                printf("\nError Kernel 1 :: %s", hipGetErrorString(error));

            // copy compressed data from GPU to CPU memory
            error =
                hipMemcpy(inputFileData, device_inputFileData,
                           ((compressedDataOffset[inputFileLength] / 8)) * sizeof(unsigned char),
                           hipMemcpyDeviceToHost);
            if (error != hipSuccess) printf("\nError 9 :: %s", hipGetErrorString(error));

            // free allocated memory
            hipFree(device_inputFileData);
            hipFree(device_compressedDataOffset);
            hipFree(device_huffmanDictionary);
            hipFree(device_byteCompressedData);
        }

        // with overflow
        else {
            // additional variable to store offset data after integer oveflow
            unsigned char *device_byteCompressedDataOverflow;

            // allocate memory to store offset information
            error =
                hipMalloc((void **)&device_byteCompressedData,
                           compressedDataOffset[integerOverflowIndex[0]] * sizeof(unsigned char));
            if (error != hipSuccess) printf("Error 10 :: %s\n", hipGetErrorString(error));

            error = hipMalloc((void **)&device_byteCompressedDataOverflow,
                               compressedDataOffset[inputFileLength] * sizeof(unsigned char));
            if (error != hipSuccess) printf("Error 11 :: %s\n", hipGetErrorString(error));

            // initialize offset data
            error =
                hipMemset(device_byteCompressedData, 0,
                           compressedDataOffset[integerOverflowIndex[0]] * sizeof(unsigned char));
            if (error != hipSuccess) printf("Error 12 :: %s\n", hipGetErrorString(error));

            error = hipMemset(device_byteCompressedDataOverflow, 0,
                               compressedDataOffset[inputFileLength] * sizeof(unsigned char));
            if (error != hipSuccess) printf("Error 13 :: %s\n", hipGetErrorString(error));

            // debug
            if (1) {
                hipMemGetInfo(&memoryFree, &memoryTotal);
                printf("Free Mem :: %lu\n", memoryFree);
            }

            // launch kernel
            compress<<<1, BLOCK_SIZE>>>(device_inputFileData, device_compressedDataOffset,
                                        device_huffmanDictionary, device_byteCompressedData,
                                        device_byteCompressedDataOverflow, inputFileLength,
                                        constMemoryFlag, integerOverflowIndex[0]);

            // check status
            hipError_t error_kernel = hipGetLastError();
            if (error_kernel != hipSuccess)
                printf("\nError Kernel 2: %s", hipGetErrorString(error_kernel));

            // get output data
            if (bitPaddingFlag[0] == 0) {
                error = hipMemcpy(
                    inputFileData, device_inputFileData,
                    (compressedDataOffset[integerOverflowIndex[0]] / 8) * sizeof(unsigned char),
                    hipMemcpyDeviceToHost);
                if (error != hipSuccess) printf("Error 14 :: %s\n", hipGetErrorString(error));

                error = hipMemcpy(
                    &inputFileData[(compressedDataOffset[integerOverflowIndex[0]] / 8)],
                    &device_inputFileData[(compressedDataOffset[integerOverflowIndex[0]] / 8)],
                    ((compressedDataOffset[inputFileLength] / 8)) * sizeof(unsigned char),
                    hipMemcpyDeviceToHost);
                if (error != hipSuccess) printf("Error 15 :: %s\n", hipGetErrorString(error));
            } else {
                error = hipMemcpy(
                    inputFileData, device_inputFileData,
                    (compressedDataOffset[integerOverflowIndex[0]] / 8) * sizeof(unsigned char),
                    hipMemcpyDeviceToHost);
                if (error != hipSuccess) printf("Error 16 :: %s\n", hipGetErrorString(error));

                unsigned char temp_compByte =
                    inputFileData[(compressedDataOffset[integerOverflowIndex[0]] / 8) - 1];

                error = hipMemcpy(
                    &inputFileData[(compressedDataOffset[integerOverflowIndex[0]] / 8) - 1],
                    &device_inputFileData[(compressedDataOffset[integerOverflowIndex[0]] / 8)],
                    ((compressedDataOffset[inputFileLength] / 8)) * sizeof(unsigned char),
                    hipMemcpyDeviceToHost);
                if (error != hipSuccess) printf("Error 17 :: %s\n", hipGetErrorString(error));

                inputFileData[(compressedDataOffset[integerOverflowIndex[0]] / 8) - 1] =
                    temp_compByte |
                    inputFileData[(compressedDataOffset[integerOverflowIndex[0]] / 8) - 1];
            }

            // free allocated memory
            hipFree(device_inputFileData);
            hipFree(device_compressedDataOffset);
            hipFree(device_huffmanDictionary);
            hipFree(device_byteCompressedData);
            hipFree(device_byteCompressedDataOverflow);
        }
    }

    // multiple run
    else {
        // no overflow
        if (integerOverflowFlag == 0) {
            error = hipMalloc(
                (void **)&device_byteCompressedData,
                (compressedDataOffset[gpuMemoryOverflowIndex[1]]) * sizeof(unsigned char));
            if (error != hipSuccess) printf("Error 18 :: %s\n", hipGetErrorString(error));

            // debug
            if (1) {
                hipMemGetInfo(&memoryFree, &memoryTotal);
                printf("\nFree Mem: %lu\n", memoryFree);
            }

            unsigned int pos = 0;
            for (unsigned int i = 0; i < numberOfKernels; i++) {
                // initialize d_byteCompressedData
                error = hipMemset(
                    device_byteCompressedData, 0,
                    compressedDataOffset[gpuMemoryOverflowIndex[1]] * sizeof(unsigned char));
                if (error != hipSuccess) printf("Error 19 :: %s\n", hipGetErrorString(error));

                compress<<<1, BLOCK_SIZE>>>(device_inputFileData, device_compressedDataOffset,
                                            device_huffmanDictionary, device_byteCompressedData,
                                            gpuMemoryOverflowIndex[i * 2], constMemoryFlag,
                                            gpuMemoryOverflowIndex[i * 2 + 1]);
                hipError_t error_kernel = hipGetLastError();
                if (error != hipSuccess) printf("Error 20 :: %s\n", hipGetErrorString(error));

                if (gpuBitPaddingFlag[i] == 0) {
                    error =
                        hipMemcpy(&inputFileData[pos], device_inputFileData,
                                   (compressedDataOffset[gpuMemoryOverflowIndex[i * 2 + 1]] / 8) *
                                       sizeof(unsigned char),
                                   hipMemcpyDeviceToHost);
                    if (error != hipSuccess) printf("Error 21 :: %s\n", hipGetErrorString(error));

                    pos += (compressedDataOffset[gpuMemoryOverflowIndex[i * 2 + 1]] / 8);
                } else {
                    unsigned char temp_compByte = inputFileData[pos - 1];
                    error =
                        hipMemcpy(&inputFileData[pos - 1], device_inputFileData,
                                   ((compressedDataOffset[gpuMemoryOverflowIndex[i * 2 + 1]] / 8)) *
                                       sizeof(unsigned char),
                                   hipMemcpyDeviceToHost);
                    if (error != hipSuccess) printf("Error 22 :: %s\n", hipGetErrorString(error));

                    inputFileData[pos - 1] = temp_compByte | inputFileData[pos - 1];
                    pos += (compressedDataOffset[gpuMemoryOverflowIndex[i * 2 + 1]] / 8) - 1;
                }
            }

            // free allocated memory
            hipFree(device_inputFileData);
            hipFree(device_compressedDataOffset);
            hipFree(device_huffmanDictionary);
            hipFree(device_byteCompressedData);
        }

        else {
            // additional variable to store offset data after integer oveflow
            unsigned char *device_byteCompressedDataOverflow;
            error =
                hipMalloc((void **)&device_byteCompressedDataOverflow,
                           (compressedDataOffset[integerOverflowIndex[0]]) * sizeof(unsigned char));
            if (error != hipSuccess) printf("Error 23 :: %s\n", hipGetErrorString(error));

            error =
                hipMalloc((void **)&device_byteCompressedDataOverflow,
                           compressedDataOffset[gpuMemoryOverflowIndex[1]] * sizeof(unsigned char));
            if (error != hipSuccess) printf("Error 22 :: %s\n", hipGetErrorString(error));

            // debug
            if (1) {
                hipMemGetInfo(&memoryFree, &memoryTotal);
                printf("Free Mem: %lu\n", memoryFree);
            }

            unsigned int pos = 0;
            for (unsigned int i = 0; i < numberOfKernels; i++) {
                if (integerOverflowIndex[i] != 0) {
                    // initialize device_byteCompressedData
                    error = hipMemset(
                        device_byteCompressedData, 0,
                        compressedDataOffset[integerOverflowIndex[0]] * sizeof(unsigned char));
                    if (error != hipSuccess) printf("Error 22 :: %s\n", hipGetErrorString(error));

                    error = hipMemset(
                        device_byteCompressedDataOverflow, 0,
                        compressedDataOffset[gpuMemoryOverflowIndex[1]] * sizeof(unsigned char));
                    if (error != hipSuccess) printf("Error 23 :: %s\n", hipGetErrorString(error));

                    compress<<<1, BLOCK_SIZE>>>(
                        device_inputFileData, device_compressedDataOffset, device_huffmanDictionary,
                        device_byteCompressedData, device_byteCompressedDataOverflow,
                        gpuMemoryOverflowIndex[i * 2], constMemoryFlag,
                        gpuMemoryOverflowIndex[i * 2 + 1], integerOverflowIndex[i]);
                    hipError_t error_kernel = hipGetLastError();
                    if (error_kernel != hipSuccess)
                        printf("Error kernel 3 :: %s\n", hipGetErrorString(error_kernel));

                    if (gpuBitPaddingFlag[i] == 0) {
                        if (bitPaddingFlag[i] == 0) {
                            error = hipMemcpy(&inputFileData[pos], device_inputFileData,
                                               (compressedDataOffset[integerOverflowIndex[i]] / 8) *
                                                   sizeof(unsigned char),
                                               hipMemcpyDeviceToHost);
                            if (error != hipSuccess)
                                printf("Error 24 :: %s\n", hipGetErrorString(error));

                            error = hipMemcpy(
                                &inputFileData[pos +
                                               (compressedDataOffset[integerOverflowIndex[i]] / 8)],
                                &device_inputFileData[(
                                    compressedDataOffset[integerOverflowIndex[i]] / 8)],
                                ((compressedDataOffset[gpuMemoryOverflowIndex[i * 2 + 1]] / 8)) *
                                    sizeof(unsigned char),
                                hipMemcpyDeviceToHost);
                            if (error != hipSuccess)
                                printf("Error 25 :: %s\n", hipGetErrorString(error));

                            pos += (compressedDataOffset[integerOverflowIndex[i]] / 8) +
                                   (compressedDataOffset[gpuMemoryOverflowIndex[i * 2 + 1]] / 8);
                        } else {
                            error = hipMemcpy(&inputFileData[pos], device_inputFileData,
                                               (compressedDataOffset[integerOverflowIndex[i]] / 8) *
                                                   sizeof(unsigned char),
                                               hipMemcpyDeviceToHost);
                            if (error != hipSuccess)
                                printf("Error 26 :: %s\n", hipGetErrorString(error));

                            unsigned char temp_compByte =
                                inputFileData[pos +
                                              (compressedDataOffset[integerOverflowIndex[i]] / 8) -
                                              1];

                            error = hipMemcpy(
                                &inputFileData[pos +
                                               (compressedDataOffset[integerOverflowIndex[i]] / 8) -
                                               1],
                                &device_inputFileData[(
                                    compressedDataOffset[integerOverflowIndex[i]] / 8)],
                                ((compressedDataOffset[gpuMemoryOverflowIndex[i * 2 + 1]] / 8)) *
                                    sizeof(unsigned char),
                                hipMemcpyDeviceToHost);
                            if (error != hipSuccess)
                                printf("Error 27 :: %s\n", hipGetErrorString(error));

                            inputFileData[pos +
                                          (compressedDataOffset[integerOverflowIndex[i]] / 8) - 1] =
                                temp_compByte |
                                inputFileData[pos +
                                              (compressedDataOffset[integerOverflowIndex[i]] / 8) -
                                              1];
                            pos += (compressedDataOffset[integerOverflowIndex[i]] / 8) +
                                   (compressedDataOffset[gpuMemoryOverflowIndex[i * 2 + 1]] / 8) -
                                   1;
                        }
                    }

                    // padding is done
                    else {
                        unsigned char temp_gpuCompByte = inputFileData[pos - 1];

                        if (bitPaddingFlag[i] == 0) {
                            error = hipMemcpy(&inputFileData[pos - 1], device_inputFileData,
                                               (compressedDataOffset[integerOverflowIndex[i]] / 8) *
                                                   sizeof(unsigned char),
                                               hipMemcpyDeviceToHost);
                            if (error != hipSuccess)
                                printf("Error 28 :: %s\n", hipGetErrorString(error));

                            error = hipMemcpy(
                                &inputFileData[pos - 1 +
                                               (compressedDataOffset[integerOverflowIndex[i]] / 8)],
                                &device_inputFileData[(
                                    compressedDataOffset[integerOverflowIndex[i]] / 8)],
                                ((compressedDataOffset[gpuMemoryOverflowIndex[i * 2 + 1]] / 8)) *
                                    sizeof(unsigned char),
                                hipMemcpyDeviceToHost);
                            if (error != hipSuccess)
                                printf("Error 29 :: %s\n", hipGetErrorString(error));

                            inputFileData[pos - 1] = temp_gpuCompByte | inputFileData[pos - 1];
                            pos += (compressedDataOffset[integerOverflowIndex[i]] / 8) +
                                   (compressedDataOffset[gpuMemoryOverflowIndex[i * 2 + 1]] / 8) -
                                   1;
                        } else {
                            error = hipMemcpy(&inputFileData[pos - 1], device_inputFileData,
                                               (compressedDataOffset[integerOverflowIndex[i]] / 8) *
                                                   sizeof(unsigned char),
                                               hipMemcpyDeviceToHost);
                            if (error != hipSuccess)
                                printf("Error 30 :: %s\n", hipGetErrorString(error));

                            unsigned char temp_compByte =
                                inputFileData[pos - 1 +
                                              (compressedDataOffset[integerOverflowIndex[i]] / 8) -
                                              1];

                            error = hipMemcpy(
                                &inputFileData[(compressedDataOffset[integerOverflowIndex[i]] / 8) -
                                               1],
                                &device_inputFileData[(
                                    compressedDataOffset[integerOverflowIndex[i]] / 8)],
                                ((compressedDataOffset[gpuMemoryOverflowIndex[i * 2 + 1]] / 8)) *
                                    sizeof(unsigned char),
                                hipMemcpyDeviceToHost);
                            if (error != hipSuccess)
                                printf("Error 31 :: %s\n", hipGetErrorString(error));

                            inputFileData[(compressedDataOffset[pos - 1 + integerOverflowIndex[i]] /
                                           8) -
                                          1] =
                                temp_compByte |
                                inputFileData[pos - 1 +
                                              (compressedDataOffset[integerOverflowIndex[i]] / 8) -
                                              1];
                            inputFileData[pos - 1] = temp_gpuCompByte | inputFileData[pos - 1];
                            pos += (compressedDataOffset[integerOverflowIndex[i]] / 8) +
                                   (compressedDataOffset[gpuMemoryOverflowIndex[i * 2 + 1]] / 8) -
                                   2;
                        }
                    }
                } else {
                    // initialize device_byteCompressedData
                    error = hipMemset(
                        device_byteCompressedData, 0,
                        compressedDataOffset[integerOverflowIndex[0]] * sizeof(unsigned char));
                    if (error != hipSuccess) printf("Error 32 :: %s\n", hipGetErrorString(error));

                    compress<<<1, BLOCK_SIZE>>>(device_inputFileData, device_compressedDataOffset,
                                                device_huffmanDictionary, device_byteCompressedData,
                                                gpuMemoryOverflowIndex[i * 2], constMemoryFlag,
                                                gpuMemoryOverflowIndex[i * 2 + 1]);
                    hipError_t error_kernel = hipGetLastError();
                    if (error_kernel != hipSuccess)
                        printf("Error Kernel 4 :: %s\n", hipGetErrorString(error_kernel));

                    if (gpuBitPaddingFlag[i] == 0) {
                        error = hipMemcpy(
                            &inputFileData[pos], device_inputFileData,
                            (compressedDataOffset[gpuMemoryOverflowIndex[i * 2 + 1]] / 8) *
                                sizeof(unsigned char),
                            hipMemcpyDeviceToHost);
                        if (error != hipSuccess)
                            printf("Error 33 :: %s\n", hipGetErrorString(error));

                        pos += (compressedDataOffset[gpuMemoryOverflowIndex[i * 2 + 1]] / 8);
                    } else {
                        unsigned char temp_huffmanTreeNode = inputFileData[pos - 1];
                        error = hipMemcpy(
                            &inputFileData[pos - 1], device_inputFileData,
                            ((compressedDataOffset[gpuMemoryOverflowIndex[i * 2 + 1]] / 8)) *
                                sizeof(unsigned char),
                            hipMemcpyDeviceToHost);
                        if (error != hipSuccess)
                            printf("Error 34 :: %s\n", hipGetErrorString(error));

                        inputFileData[pos - 1] = temp_huffmanTreeNode | inputFileData[pos - 1];
                        pos += (compressedDataOffset[gpuMemoryOverflowIndex[i * 2 + 1]] / 8) - 1;
                    }
                }
            }

            // free allocated memory
            hipFree(device_inputFileData);
            hipFree(device_compressedDataOffset);
            hipFree(device_huffmanDictionary);
            hipFree(device_byteCompressedData);
        }
    }
}
