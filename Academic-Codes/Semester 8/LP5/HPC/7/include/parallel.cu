#include "hip/hip_runtime.h"
/*
 * Parallel function implementations
 * nvcc supported file won't compile with gcc
 */

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "parallel.h"

/*
 * Sorting the nodes based on the frequency
 * The man frequency is represented by the distinct char count
 */
void sortHuffmanTree(int a, int distinctCharacterCount, int combinedHuffmanNodes) {
    for (int i = combinedHuffmanNodes; i < distinctCharacterCount - 1 + a; i++) {
        for (int j = combinedHuffmanNodes; j < distinctCharacterCount - 1 + a; j++) {
            // perform swapping
            if (huffmanTreeNode[j].frequency > huffmanTreeNode[j + 1].frequency) {
                struct huffmanNode tempNode = huffmanTreeNode[j];
                huffmanTreeNode[j] = huffmanTreeNode[j + 1];
                huffmanTreeNode[j + 1] = tempNode;
            }
        }
    }
}

/*
 * Build the tree from the sorted results
 * The tree here is the min heap
 */
void buildHuffmanTree(int i, int distinctCharacterCount, int combinedHuffmanNodes) {
    huffmanTreeNode[distinctCharacterCount + i].frequency =
        huffmanTreeNode[combinedHuffmanNodes].frequency +
        huffmanTreeNode[combinedHuffmanNodes + 1].frequency;
    huffmanTreeNode[distinctCharacterCount + i].left = &huffmanTreeNode[combinedHuffmanNodes];
    huffmanTreeNode[distinctCharacterCount + i].right = &huffmanTreeNode[combinedHuffmanNodes + 1];
    huffmanTreeNode_head = &(huffmanTreeNode[distinctCharacterCount + i]);
}

/*
 * Build the dictionary for the huffman tree
 * It will store the bit sequence and their respective lengths
 */
void buildHuffmanDictionary(struct huffmanNode* root, unsigned char* bitSequence,
                            unsigned char bitSequenceLength) {
    if (root->left) {
        bitSequence[bitSequenceLength] = 0;
        buildHuffmanDictionary(root->left, bitSequence, bitSequenceLength + 1);
    }

    if (root->right) {
        bitSequence[bitSequenceLength] = 1;
        buildHuffmanDictionary(root->right, bitSequence, bitSequenceLength + 1);
    }

    // copy the bit sequence and the length to the dictionary
    if (root->right == NULL && root->left == NULL) {
        huffmanDictionary.bitSequenceLength[root->letter] = bitSequenceLength;
        if (bitSequenceLength < 192) {
            memcpy(huffmanDictionary.bitSequence[root->letter], bitSequence,
                   bitSequenceLength * sizeof(unsigned char));
        } else {
            memcpy(bitSequenceConstMemory[root->letter], bitSequence,
                   bitSequenceLength * sizeof(unsigned char));
            memcpy(huffmanDictionary.bitSequence[root->letter], bitSequence, 191);
            constMemoryFlag = 1;
        }
    }
}

/*
 * Generate data offset array
 * Case :- Single run, no overflow
 */
void createDataOffsetArray(unsigned int* compressedDataOffset, unsigned char* inputFileData,
                           unsigned int inputFileLength) {
    compressedDataOffset[0] = 0;
    for (int i = 0; i < inputFileLength; i++) {
        compressedDataOffset[i + 1] =
            huffmanDictionary.bitSequenceLength[inputFileData[i]] + compressedDataOffset[i];
    }
    // not a byte & remaining values
    if (compressedDataOffset[inputFileLength] % 8 != 0) {
        compressedDataOffset[inputFileLength] = compressedDataOffset[inputFileLength] +
                                                (8 - (compressedDataOffset[inputFileLength] % 8));
    }
}

/*
 * Generate data offset array
 * Case :- Single run, with overflow
 * note : calculate compressed data offset - (1048576 is a safe number that will ensure there is no
 * integer overflow in GPU, it should be minimum 8 * number of threads)
 */
void createDataOffsetArray(unsigned int* compressedDataOffset, unsigned char* inputFileData,
                           unsigned int inputFileLength, unsigned int* integerOverflowIndex,
                           unsigned int* bitPaddingFlag, int numBytes) {
    int j = 0;
    compressedDataOffset[0] = 0;

    for (int i = 0; i < inputFileLength; i++) {
        compressedDataOffset[i + 1] =
            huffmanDictionary.bitSequenceLength[inputFileData[i]] + compressedDataOffset[i];

        if (compressedDataOffset[i + 1] + numBytes < compressedDataOffset[i]) {
            integerOverflowIndex[j] = i;

            if (compressedDataOffset[j] % 8 != 0) {
                bitPaddingFlag[j] = 1;
                compressedDataOffset[i + 1] = (compressedDataOffset[i] % 8) +
                                              huffmanDictionary.bitSequenceLength[inputFileData[i]];
                compressedDataOffset[i] =
                    compressedDataOffset[i] + (8 - (compressedDataOffset[i] % 8));
            } else {
                compressedDataOffset[i + 1] = huffmanDictionary.bitSequenceLength[inputFileData[i]];
            }
            j++;
        }
    }

    if (compressedDataOffset[inputFileLength] % 8 != 0) {
        compressedDataOffset[inputFileLength] = compressedDataOffset[inputFileLength] +
                                                (8 - (compressedDataOffset[inputFileLength] % 8));
    }
}

/*
 * Generate data offset array
 * Case :- Multiple run, with no overflow
 */
void createDataOffsetArray(unsigned int* compressedDataOffset, unsigned char* inputFileData,
                           unsigned int inputFileLength, unsigned int* gpuMemoryOverflow,
                           unsigned int* gpuBitPaddingFlag, long unsigned int memoryRequired) {
    int j = 0;
    gpuMemoryOverflow[0] = 0;
    gpuBitPaddingFlag[0] = 0;
    compressedDataOffset[0] = 0;

    for (int i = 0; i < inputFileLength; i++) {
        compressedDataOffset[i + 1] =
            huffmanDictionary.bitSequenceLength[inputFileData[i]] + compressedDataOffset[i];

        if (compressedDataOffset[i + 1] > memoryRequired) {
            gpuMemoryOverflow[j * 2 + 1] = i;
            gpuMemoryOverflow[j * 2 + 2] = i + 1;

            if (compressedDataOffset[i] % 8 != 0) {
                gpuBitPaddingFlag[j + 1] = 1;
                compressedDataOffset[i + 1] = (compressedDataOffset[i] % 8) +
                                              huffmanDictionary.bitSequenceLength[inputFileData[i]];
                compressedDataOffset[i] =
                    compressedDataOffset[i] + (8 - (compressedDataOffset[i] % 8));
            } else {
                compressedDataOffset[i + 1] = huffmanDictionary.bitSequenceLength[inputFileData[i]];
            }

            j++;
        }
    }

    if (compressedDataOffset[inputFileLength] % 8 != 0) {
        compressedDataOffset[inputFileLength] = compressedDataOffset[inputFileLength] +
                                                (8 - (compressedDataOffset[inputFileLength] % 8));
    }

    gpuMemoryOverflow[j * 2 + 1] = inputFileLength;
}

/*
 * Generate data offset array
 * Case :- Multiple run, with overflow
 */
void createDataOffsetArray(unsigned int* compressedDataOffset, unsigned char* inputFileData,
                           unsigned int inputFileLength, unsigned int* integerOverflowIndex,
                           unsigned int* bitPaddingFlag, unsigned int* gpuMemoryOverflowIndex,
                           unsigned int* gpuBitPaddingFlag, int numBytes,
                           long unsigned int memoryRequired) {
    int j = 0, k = 0;
    compressedDataOffset[0] = 0;

    for (int i = 0; i < inputFileLength; i++) {
        compressedDataOffset[i + 1] =
            huffmanDictionary.bitSequenceLength[inputFileData[i]] + compressedDataOffset[i];
        if (j != 0 && (long unsigned int)compressedDataOffset[i + 1] +
                              compressedDataOffset[integerOverflowIndex[j - 1]] >
                          memoryRequired) {
            gpuMemoryOverflowIndex[k * 2 + 1] = i;
            gpuMemoryOverflowIndex[k * 2 + 2] = i + 1;

            if (compressedDataOffset[i] % 8 != 0) {
                gpuBitPaddingFlag[k + 1] = 1;
                compressedDataOffset[i + 1] = (compressedDataOffset[i] % 8) +
                                              huffmanDictionary.bitSequenceLength[inputFileData[i]];
                compressedDataOffset[i] =
                    compressedDataOffset[i] + (8 - (compressedDataOffset[i] % 8));
            } else {
                compressedDataOffset[i + 1] = huffmanDictionary.bitSequenceLength[inputFileData[i]];
            }

            k++;
        } else if (compressedDataOffset[i + 1] + numBytes < compressedDataOffset[i]) {
            integerOverflowIndex[j] = i;

            // if not a byte
            if (compressedDataOffset[i] % 8 != 0) {
                bitPaddingFlag[j] = 1;
                compressedDataOffset[i + 1] = (compressedDataOffset[i] % 8) +
                                              huffmanDictionary.bitSequenceLength[inputFileData[i]];
                compressedDataOffset[i] =
                    compressedDataOffset[i] + (8 - (compressedDataOffset[i] % 8));
            } else {
                compressedDataOffset[i + 1] = huffmanDictionary.bitSequenceLength[inputFileData[i]];
            }

            j++;
        }
    }

    // remaining values
    if (compressedDataOffset[inputFileLength] % 8 != 0) {
        compressedDataOffset[inputFileLength] = compressedDataOffset[inputFileLength] +
                                                (8 - (compressedDataOffset[inputFileLength] % 8));
    }

    gpuMemoryOverflowIndex[j * 2 + 1] = inputFileLength;
}
